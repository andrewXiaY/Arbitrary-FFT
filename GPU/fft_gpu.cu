#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cmath>
#include <iostream>
#include <random>
#include <type_traits>
#include <vector>
#include <stdio.h>
#include <omp.h>
#include "kernels.cuh"
#include <assert.h>
#include <stdlib.h>
#include <math.h>

// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// Helper functions for CUDA
#include ""

struct NaiveDFT_cuda {
    static constexpr char Name[] = "NaiveDFT_cuda";
    const std::size_t N;

    NaiveDFT_cuda(std::size_t N) : N(N) {}

    ~NaiveDFT_cuda(){hipDeviceReset();}

    void dft(Comp* Y, const Comp* X){
      using namespace std;

      size_t xblock = std::sqrt(BLOCKSIZE); // currently support 2^n
      dim3 dimBlock(xblock, xblock); // define the size of block
      
      size_t xgrid = (N + dimBlock.x - 1) / dimBlock.x;
      size_t ygrid = (N + dimBlock.y - 1) / dimBlock.y;
      dim3 dimGrid(xgrid, ygrid); 

      // initialize host and device of X
      hipDoubleComplex* h_X = (hipDoubleComplex*) malloc(N*sizeof(hipDoubleComplex));  // allocate memory to host
      Comp_to_cuComp(X, h_X, N);  // convert Comp to cuda complex
      hipDoubleComplex* d_X;
      hipMalloc(&d_X, N*sizeof(hipDoubleComplex));
      hipMemcpy(d_X, h_X, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

      // initialize host and device of Y
      hipDoubleComplex* h_Y = (hipDoubleComplex*) malloc(N*ygrid*sizeof(hipDoubleComplex));
      hipDoubleComplex* d_Y;
      hipMalloc(&d_Y, N*ygrid*sizeof(hipDoubleComplex));
      for(int i = 0; i < N*ygrid; ++i)
        h_Y[i] = make_hipDoubleComplex(0.0, 0.0);
      hipMemcpy(d_Y, h_Y, N*ygrid*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
      hipDeviceSynchronize();

      double tt = omp_get_wtime();
      dft_kernel2d<<<dimGrid, dimBlock>>>(d_X, d_Y, N, ygrid);
      Check_CUDA_Error("Error");
      reduction_2d<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_Y, N, ygrid);
      Check_CUDA_Error("Error");
      hipDeviceSynchronize();
      hipMemcpy(h_Y, d_Y, N*ygrid*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

      for(int i = 0; i < N; ++i)
        Y[i] = Comp(hipCreal(h_Y[i*ygrid]), hipCimag(h_Y[i*ygrid]));

        // cout << "[" << Name  << "] (dft) run time: " <<  omp_get_wtime() - tt << endl;
      free(h_Y);
      free(h_X);
    }

    void idft(Comp* Y, const Comp* X) {
      using namespace std;

      size_t xblock = std::sqrt(BLOCKSIZE); // currently support 2^n
      dim3 dimBlock(xblock, xblock); // define the size of block
      
      size_t xgrid = (N + dimBlock.x - 1) / dimBlock.x;
      size_t ygrid = (N + dimBlock.y - 1) / dimBlock.y;
      dim3 dimGrid(xgrid, ygrid); 

      // initialize host and device of X
      hipDoubleComplex* h_X = (hipDoubleComplex*) malloc(N*sizeof(hipDoubleComplex));  // allocate memory to host
      Comp_to_cuComp(X, h_X, N);  // convert Comp to cuda complex
      hipDoubleComplex* d_X;
      hipMalloc(&d_X, N*sizeof(hipDoubleComplex));
      hipMemcpy(d_X, h_X, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

      // initialize host and device of Y
      hipDoubleComplex* h_Y = (hipDoubleComplex*) malloc(N*ygrid*sizeof(hipDoubleComplex));
      hipDoubleComplex* d_Y;
      hipMalloc(&d_Y, N*ygrid*sizeof(hipDoubleComplex));
      for(int i = 0; i < N*ygrid; ++i)
        h_Y[i] = make_hipDoubleComplex(0.0, 0.0);
      hipMemcpy(d_Y, h_Y, N*ygrid*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
      hipDeviceSynchronize();

      double tt = omp_get_wtime();
      idft_kernel2d<<<dimGrid, dimBlock>>>(d_X, d_Y, N, ygrid);
      Check_CUDA_Error("Error");
      reduction_2d<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_Y, N, ygrid);
      Check_CUDA_Error("Error");
      hipDeviceSynchronize();
      hipMemcpy(h_Y, d_Y, N*ygrid*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
  
      for(int i = 0; i < N; ++i)
        Y[i] = Comp(hipCreal(h_Y[i*ygrid]) / N, hipCimag(h_Y[i*ygrid]) / N);
    cout << "[" << Name  << "] (idft) run time: " <<  omp_get_wtime() - tt << endl;

      free(h_Y);
      free(h_X);
    }
};

// Decimation in Time FFT
// Performs bit-reverse-copy before transform
// N must be power of 2
struct DitFFT_cuda {
    static constexpr char Name[] = "DitFFT_cuda";
    const std::size_t N;
    const int L;
    hipDoubleComplex* cuda_W;
    Comp* W;
    hipDoubleComplex* d_X;
    hipDoubleComplex* d_Y;
    size_t* d_I;
    std::size_t* I;

    DitFFT_cuda(std::size_t N) : N(N), L(__builtin_ctzll(N)) {
        using namespace std;
        // initialize W
        auto w = _2Pi / (Real) N;

        // allocate memory 
        hipMalloc(&cuda_W, N*sizeof(hipDoubleComplex));
        hipMalloc(&d_X, N*sizeof(hipDoubleComplex));
        hipMalloc(&d_Y, N*sizeof(hipDoubleComplex));
        hipMalloc(&d_I, N*sizeof(size_t));

        I = (std::size_t*) malloc(N*sizeof(std::size_t));
        W = (Comp*) malloc(N*sizeof(Comp));

        init_w_kernel<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(cuda_W, w, N);

        // initialize I
        
        for (size_t i = 0, j = 0; i < N; ++i) {
            I[i] = j;
            for (size_t k = N >> 1; (j ^= k) < k; k >>= 1);
        }

        hipMemcpy(d_I, I, N*sizeof(size_t), hipMemcpyHostToDevice);

        // hipMemcpy(W, cuda_W, N*sizeof(size_t), hipMemcpyDeviceToHost);
        for (size_t i = 0; i < N; ++i)
            W[i] = exp(Comp(0, w * (Real) i));
    }

    ~DitFFT_cuda(){
        free(W);
        free(I);
        hipFree(d_X);
        hipFree(d_Y);
        hipFree(cuda_W);
        hipFree(d_I);
    }

    void dft(Comp* Y, const Comp* X) const {
        using namespace std;
        hipMemcpy(d_X, X, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(d_Y, Y, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        
        bitrev_kernel<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_Y, d_X, d_I, N);

        for(int i = 0; i < L; ++i){
            auto h = size_t{1} << i;
            dim3 blockDim(h < 1024 ? (BLOCKSIZE + h - 1) / h : 1, h < 1024 ? h : 1024);
            dim3 gridDim((N/(2*h) + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
            ditfft_kernel<<<gridDim, blockDim>>>(d_Y, cuda_W, i, N);
        }
        
        hipMemcpy(Y, d_Y, N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    }

    void idft(Comp* Y, const Comp* X) const {

        using namespace std;
        hipMemcpy(d_X, X, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(d_Y, Y, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        
        bitrev_kernel<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_Y, d_X, d_I, N);

        double tt = omp_get_wtime();
        for(int i = 0; i < L; ++i){
            auto h = size_t{1} << i;
            dim3 blockDim(h < 1024 ? (BLOCKSIZE + h - 1) / h : 1, h < 1024 ? h : 1024);
            dim3 gridDim((N/(2*h) + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
            ditdifidft_kernel<<<gridDim, blockDim>>>(d_Y, cuda_W, i, N);
            Check_CUDA_Error("267");
        }

        complex_div_real<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_Y, N);
        hipMemcpy(Y, d_Y, N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    }
};


// Forward transform uses DIF while inverse transform uses DIT
// For convolution use, no bit-reverse-copy performed, performed in-place
// N must be power of 2
struct DifDitFFT_cuda {
    static constexpr char Name[] = "DifDitFFT_cuda";
    const std::size_t N;
    const int L;
    hipDoubleComplex* cuda_W;
    hipDoubleComplex* d_Y;
    Comp* W;

    DifDitFFT_cuda(std::size_t N) : N(N), L(__builtin_ctzll(N)) {
        using namespace std;
        auto w = _2Pi / (Real) N;

        hipMalloc(&cuda_W, N*sizeof(hipDoubleComplex));
        hipMalloc(&d_Y, N*sizeof(hipDoubleComplex));
        W = (Comp*) malloc(N*sizeof(Comp));

        init_w_kernel<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(cuda_W, w, N);
        hipDeviceSynchronize();
        hipMemcpy(W, cuda_W, N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    }

    ~DifDitFFT_cuda(){
        free(W);
        hipFree(cuda_W);
        hipFree(d_Y);
    }

    void dft(Comp* Y) {
        using namespace std;

        hipMemcpy(d_Y, Y, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        double tt = omp_get_wtime();
        for(int i = L - 1; i >= 0; --i){
            auto h = size_t{1} << i;
            dim3 blockDim(h < 1024 ? (BLOCKSIZE + h - 1) / h : 1, h < 1024 ? h : 1024);
            dim3 gridDim((N/(2*h) + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
            ditdiffft_kernel<<<gridDim, blockDim>>>(d_Y, cuda_W, i, N);
        }

        hipMemcpy(Y, d_Y, N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        Check_CUDA_Error("395");
        // cout << "[" << Name  << "] (dft) run time: " <<  omp_get_wtime() - tt << endl;
    }

    void idft(Comp* Y) {
        using namespace std;

        hipMemcpy(d_Y, Y, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

        double tt = omp_get_wtime();
        for(int i = 0; i < L; ++i){
            auto h = size_t{1} << i;
            dim3 blockDim(h < 1024 ? (BLOCKSIZE + h - 1) / h : 1, h < 1024 ? h : 1024);
            dim3 gridDim((N/(2*h) + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
            ditdifidft_kernel<<<gridDim, blockDim>>>(d_Y, cuda_W, i, N);
        }
        complex_div_real<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_Y, N);
        hipMemcpy(Y, d_Y, N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
        Check_CUDA_Error("406");
        // cout << "[" << Name  << "] (idft) run time: " <<  omp_get_wtime() - tt << endl;
    }
};

template<class I>
constexpr I ceil2(I x) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    if (sizeof(I) > 1)
        x |= x >> 8;
    if (sizeof(I) > 2)
        x |= x >> 16;
    if (sizeof(I) > 4)
        x |= x >> 32;
    return x + 1;
}

template<class I>
constexpr bool isPow2(I x) { return ((x ^ (x - 1)) >> 1) == x - 1; }

// cuda bluesteinFFT
struct BluesteinFFT_cuda {
    static constexpr char Name[] = "BluesteinFFT_cuda";
    const std::size_t N;
    const bool pow2;
    Comp* A; 
    Comp* B; 
    Comp* C;
    hipDoubleComplex* cuda_C;
    hipDoubleComplex* d_X;
    hipDoubleComplex* d_Y;
    hipDoubleComplex* d_A;
    hipDoubleComplex* d_B;
    double tt, run_time = 0;

    union {
        char c;
        DitFFT_cuda dit;
        DifDitFFT_cuda difdit;
    };

    BluesteinFFT_cuda(std::size_t N) : N(N), pow2(isPow2(N)) {
        using namespace std;
        if (pow2) {
            new(&dit) DitFFT_cuda(N);
            return;
        }

        new(&difdit) DifDitFFT_cuda(ceil2(N + N - 1));
        auto w = Pi / (Real) N;

        // allocate memory in GPU
        hipMalloc(&d_X, N*sizeof(hipDoubleComplex));
        hipMalloc(&d_Y, N*sizeof(hipDoubleComplex));
        hipMalloc(&d_A, difdit.N*sizeof(hipDoubleComplex));
        hipMalloc(&d_B, difdit.N*sizeof(hipDoubleComplex));
        hipMalloc(&cuda_C, N*sizeof(hipDoubleComplex));

        // allocate memory for computation
        A = (Comp*) malloc(difdit.N*sizeof(Comp));
        B = (Comp*) malloc(difdit.N*sizeof(Comp));
        C = (Comp*) malloc(N*sizeof(Comp));
        
        // initialize C (W)
        init_C_kernel<<<(N + BLOCKSIZE - 1)/ BLOCKSIZE, BLOCKSIZE>>>(cuda_C, w, N);
        hipMemcpy(C, cuda_C, N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

        B[0] = C[0];

        for (size_t i = 1; i < N; ++i){
            B[i] = B[difdit.N - i] = C[i];
        }
        for(size_t i = N; i < difdit.N - N + 1; ++i){
            B[i] = Comp{};
        }
        difdit.dft(B);
        hipMemcpy(d_B, B, difdit.N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    }

    ~BluesteinFFT_cuda() {
        if (pow2) {
            dit.~DitFFT_cuda();
            return;
        }
        difdit.~DifDitFFT_cuda();
        hipDeviceReset();
    }

    void dft(Comp* Y, const Comp* X) {
        if (pow2) {
            dit.dft(Y, X);
            return;
        }
        using namespace std;

        hipMemcpy(d_X, X, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(d_Y, Y, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

        
        tt = omp_get_wtime();
        complex_mul_conj_kernel<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_A, d_X, cuda_C, N, difdit.N);

        // perform dft on A
        for(int i = difdit.L - 1; i >= 0; --i){
            auto h = size_t{1} << i;
            dim3 blockDim(h < 1024 ? (BLOCKSIZE + h - 1) / h : 1, h < 1024 ? h : 1024);
            dim3 gridDim((difdit.N/(2*h) + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
            ditdiffft_kernel<<<gridDim, blockDim>>>(d_A, difdit.cuda_W, i, difdit.N);
        }
        
        complex_self_mul_kernel<<<(difdit.N + BLOCKSIZE) / BLOCKSIZE, BLOCKSIZE>>>(d_A, d_B, difdit.N);
        
        // perform idft on A
        for(int i = 0; i < difdit.L; ++i){
            auto h = size_t{1} << i;
            dim3 blockDim(h < 1024 ? (BLOCKSIZE + h - 1) / h : 1, h < 1024 ? h : 1024);
            dim3 gridDim((difdit.N/(2*h) + blockDim.x - 1) / blockDim.x, (h + blockDim.y - 1) / blockDim.y);
            ditdifidft_kernel<<<gridDim, blockDim>>>(d_A, difdit.cuda_W, i, difdit.N);
        }
        
        complex_div_real_mul_conj<<<(difdit.N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_A, d_Y, cuda_C, difdit.N, N);

        hipDeviceSynchronize();
        
        run_time += omp_get_wtime() - tt;
        tt = omp_get_wtime();

        hipMemcpy(Y, d_Y, N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    }

    void idft(Comp* Y, const Comp* X) {
        if (pow2) {
            dit.idft(Y, X);
            return;
        }
        using namespace std;

        hipMemcpy(d_X, X, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
        hipMemcpy(d_Y, Y, N*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);


        complex_mul_kernel<<<(N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_A, d_X, cuda_C, N, difdit.N);
    
        dim3 blockDim(32, 32);
        for(int i = difdit.L - 1; i >= 0; --i){
            auto h = size_t{1} << i;
            dim3 gridDim((difdit.N/(2*h) + blockDim.x - 1) / blockDim.x, (h + blockDim.x - 1) / blockDim.x);
            ditdiffft_kernel<<<gridDim, blockDim>>>(d_A, difdit.cuda_W, i, difdit.N);
        }

        complex_self_conj_mul_kernel<<<(difdit.N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_A, d_B, difdit.N);

        for(int i = 0; i < difdit.L; ++i){
            auto h = size_t{1} << i;
            dim3 gridDim((difdit.N/(2*h) + blockDim.x - 1) / blockDim.x, (h + blockDim.x - 1) / blockDim.x);
            ditdifidft_kernel<<<gridDim, blockDim>>>(d_A, difdit.cuda_W, i, difdit.N);
        }
        complex_div_real_mul<<<(difdit.N + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE>>>(d_A, d_Y, cuda_C, difdit.N, N);
        hipMemcpy(Y, d_Y, N*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    }
};

Real error(const Comp* A, const Comp* B, std::size_t N) {
    using namespace std;
    Real res = 0;
    for (size_t i = 0; i < N; ++i)
        res = max(res, abs(A[i] - B[i]));
    return res;
}

std::mt19937_64 R(std::random_device{}());

Real randReal(Real lo, Real up) {
    return std::uniform_real_distribution<Real>(lo, up)(R);
}

Comp randComp(Real lo, Real up) {
    return {randReal(lo, up), randReal(lo, up)};
}

void fft_cuda(std::size_t sign, std::size_t N, Comp* input, Comp* output){
    using namespace std;
    BluesteinFFT_cuda bluestein_cuda(N);


    if(sign == 1){
        bluestein_cuda.dft(output, input);
    }else{
        bluestein_cuda.idft(output, input);
    }
    cout << "Blustein cuda Run time: " << bluestein_cuda.run_time << endl;
}

void print_vector(Comp* x, int N){
    for(int i = 0; i < N; ++i)
        std::cout << x[i] << std::endl;
    std::cout << std::endl;
}

void benchmark(Comp* Data, int LENGTH){
    
    hipfftDoubleComplex* CompData = (hipfftDoubleComplex*) malloc(LENGTH*sizeof(hipfftDoubleComplex));
    for(int i = 0; i < LENGTH; ++i){
        CompData[i].x = real(Data[i]);
        CompData[i].y = imag(Data[i]);
    }

    hipfftDoubleComplex *d_fftData;
    hipMalloc((void**)&d_fftData, LENGTH * sizeof(hipfftDoubleComplex));// allocate memory for the data in device
    hipMemcpy(d_fftData, CompData, LENGTH * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);// copy data from host to device

    hipfftHandle plan;// cuda library function handle
    hipfftPlan1d(&plan, LENGTH, HIPFFT_Z2Z, 1);//declaration
    
    double tt = omp_get_wtime();
    hipfftExecZ2Z(plan, (hipfftDoubleComplex*)d_fftData, (hipfftDoubleComplex*)d_fftData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done
    std::cout << "Bechmark run time: " << omp_get_wtime() - tt << std::endl;

    hipMemcpy(CompData, d_fftData, LENGTH * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);// copy the result from device to host

    Check_CUDA_Error("error");

    hipfftDestroy(plan);
    hipFree(d_fftData);
    for(int i = 0; i < LENGTH; ++i)
        Data[i] = Comp(CompData[i].x, CompData[i].y);

    free(CompData);
    
}


int main(int argc, char* argv[]){
    using namespace std;
    int N = atoi(argv[1]);
    
    Comp* input = (Comp*) malloc(N*sizeof(Comp));
    Comp* output = (Comp*) malloc(N*sizeof(Comp));
    Comp* original = (Comp*) malloc(N*sizeof(Comp));

    for(int i = 0; i < N; ++i)
        original[i] = input[i] = randComp(-10, 10);

    NaiveDFT_cuda naive_cuda(N);
    fft_cuda(1, N, input, output);
    
    benchmark(input, N);
    cout << "Error: " << error(output, input, N) << endl;

    free(input);
    free(output);
    free(original);

    return 0;
}



