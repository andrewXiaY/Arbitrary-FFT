#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <omp.h>
#include <random>
#include <iostream>
// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// Helper functions for CUDA


#define pi 3.1415926535
// #define LENGTH 1024*1024 //signal sampling points

std::mt19937_64 R(std::random_device{}());
double randReal(double lo, double up) {
    return std::uniform_real_distribution<double>(lo, up)(R);
}

int main(int argc, char* argv[])
{
    // data gen
    int LENGTH = atoi(argv[1]);
    double* Data = (double*) malloc(LENGTH*sizeof(double));
    for(int i = 0; i < LENGTH; ++i){
        Data[i] = randReal(-100, 100);
    }
    
    double fs = 1000000.000;//sampling frequency
    double f0 = 200000.00;// signal frequency

    hipfftComplex *CompData = (hipfftComplex*)malloc(LENGTH * sizeof(hipfftComplex));//allocate memory for the data in host
    int i;
    for (i = 0; i < LENGTH; i++)
    {
        CompData[i].x = Data[i];
        CompData[i].y = 0;
    }

    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData, LENGTH * sizeof(hipfftComplex));// allocate memory for the data in device
    hipMemcpy(d_fftData, CompData, LENGTH * sizeof(hipfftComplex), hipMemcpyHostToDevice);// copy data from host to device

    hipfftHandle plan;// cuda library function handle
    hipfftPlan1d(&plan, LENGTH, HIPFFT_C2C, 1);//declaration
    double tt = omp_get_wtime();
    hipfftExecC2C(plan, (hipfftComplex*)d_fftData, (hipfftComplex*)d_fftData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done
    hipMemcpy(CompData, d_fftData, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);// copy the result from device to host

    std::cout << omp_get_wtime() - tt << std::endl;

    hipfftDestroy(plan);
    free(CompData);
    hipFree(d_fftData);

}