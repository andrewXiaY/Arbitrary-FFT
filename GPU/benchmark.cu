#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <omp.h>
#include <random>
#include <iostream>
// Include CUDA runtime and CUFFT
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

// Helper functions for CUDA


#define pi 3.1415926535
#define LENGTH 1024*1024*16 //signal sampling points

std::mt19937_64 R(std::random_device{}());
double randReal(double lo, double up) {
    return std::uniform_real_distribution<double>(lo, up)(R);
}

int main()
{
    // data gen
    double* Data = (double*) malloc(LENGTH*sizeof(double));
    for(int i = 0; i < LENGTH; ++i){
        Data[i] = randReal(-100, 100);
    }
    
    double fs = 1000000.000;//sampling frequency
    double f0 = 200000.00;// signal frequency
    // for (int i = 0; i < LENGTH; i++)
    // {
    //     Data[i] = 1.35*cos(2 * pi*f0*i / fs);//signal gen,
    // }

    hipfftComplex *CompData = (hipfftComplex*)malloc(LENGTH * sizeof(hipfftComplex));//allocate memory for the data in host
    int i;
    for (i = 0; i < LENGTH; i++)
    {
        CompData[i].x = Data[i];
        CompData[i].y = 0;
    }

    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData, LENGTH * sizeof(hipfftComplex));// allocate memory for the data in device
    hipMemcpy(d_fftData, CompData, LENGTH * sizeof(hipfftComplex), hipMemcpyHostToDevice);// copy data from host to device

    hipfftHandle plan;// cuda library function handle
    hipfftPlan1d(&plan, LENGTH, HIPFFT_C2C, 1);//declaration
    double tt = omp_get_wtime();
    hipfftExecC2C(plan, (hipfftComplex*)d_fftData, (hipfftComplex*)d_fftData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done
    hipMemcpy(CompData, d_fftData, LENGTH * sizeof(hipfftComplex), hipMemcpyDeviceToHost);// copy the result from device to host

    std::cout << omp_get_wtime() - tt << std::endl;
    // for (i = 0; i < LENGTH / 2; i++)
    // {
    //     printf("i=%d\tf= %6.1fHz\tRealAmp=%3.1f\t", i, fs*i / LENGTH, CompData[i].x*2.0 / LENGTH);
    //     printf("ImagAmp=+%3.1fi", CompData[i].y*2.0 / LENGTH);
    //     printf("\n");
    // }
    hipfftDestroy(plan);
    free(CompData);
    hipFree(d_fftData);

}